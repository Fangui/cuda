
#include <hip/hip_runtime.h>
#include <iostream>

const int lines = 1024;
const int cols = 1024;
const int block_size = 16;

__global__ void 
transpose_matrix(int *input, int *output)
{
    int i = blockIdx.x * block_size;
    int j = blockIdx.y * block_size;

    int x = threadIdx.x;
    int y = threadIdx.y;

    __shared__ int block_tr[block_size * block_size];

    block_tr[y * block_size + x] = input[i + x + (j + y) * lines]; 

    __syncthreads();
    output[j + x + (i + y) * lines] = block_tr[x * block_size + y];
}

void transpose_ref(int *input, int *output)
{
    for (int i = 0; i < lines; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            output[j * lines + i] = input[i * cols + j];
        }
    }
}

int check_transpose(int *a, int *b)
{
    for (int i = 0; i < lines * cols; ++i)
    {
        if (a[i] != b[i])
        {
            std::cerr << "error while transpose" << std::endl;
            return 1;
        }
    }
    return 0;
}

int main()
{   
    int *mat_a = new int[lines * cols];
    int *mat_b = new int[lines * cols];
    std::size_t nb_bits = lines * cols * sizeof(int);

    dim3 blocks(lines / block_size, cols / block_size);
    dim3 thread(block_size, block_size);

    #pragma omp simd
    for (int i = 0; i < lines * cols; ++i)
        mat_a[i] = i;

    int *d_in;
    int *d_out;
    hipMalloc(&d_in, nb_bits);
    hipMalloc(&d_out, nb_bits);

    hipMemcpy(d_in, mat_a, nb_bits, hipMemcpyHostToDevice);

    transpose_matrix <<<blocks, thread>>> (d_in, d_out);
    hipDeviceSynchronize();

    hipMemcpy(mat_b, d_out, nb_bits, hipMemcpyDeviceToHost); 

    hipFree(d_in);
    hipFree(d_out);

    int *mat_ref = new int[lines * cols];
    transpose_ref(mat_b, mat_ref);
  
    int ret = check_transpose(mat_a, mat_ref);

    delete[] mat_a;
    delete[] mat_b;
    delete[] mat_ref;

    return ret;
}
