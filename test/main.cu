
#include <hip/hip_runtime.h>
#define SIZE     60000000
#define BLOCKDIM 1024

struct Test
{
    Test(unsigned size)
    {
        data = new int[size];
        this->size = size;
    }

    ~Test()
    {
        delete[] data;
    }

    int *data;
    unsigned size;
};

struct Pool
{
    Pool(unsigned size)
    {
        hipMalloc(&d_test, sizeof(struct Test));
        hipMalloc(&d_data, size * sizeof(int));
        hipMemcpy(&(d_test->data), &d_data, sizeof(int *), hipMemcpyHostToDevice);
        hipMemcpy(&(d_test->size), &(size), sizeof(unsigned), hipMemcpyHostToDevice);
    }
    ~Pool()
    {
        hipFree(d_test);
        hipFree(d_data);
    }
    Test *d_test;
    int *d_data;
};

__global__ void init(struct Test *t)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= t->size)
      return;

    t->data[idx] = idx;
}

void initTest()
{
    Test t(SIZE);

    unsigned size = (SIZE + BLOCKDIM - 1) / BLOCKDIM;

    static Pool p(SIZE);

    init<<<size, BLOCKDIM>>> (p.d_test);

    hipMemcpy(t.data, p.d_data, SIZE * sizeof(int), hipMemcpyDeviceToHost);
}


int main()
{
    for (unsigned i = 0; i < 10; ++i)
        initTest();

    return 0;
}
