
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 60

struct Test
{
    Test()
    {
        data = new int[SIZE];
        size = SIZE;
    }

    ~Test()
    {
        delete[] data;
    }

    int *data;
    unsigned size;
};

__global__ void init(struct Test *t)
{
    t->data[threadIdx.x] = threadIdx.x;
}

void initTest()
{
    Test *t = new Test;

    Test *d_test;
    int *d_tmp;

    hipMalloc(&d_test, sizeof(struct Test));
    hipMalloc(&d_tmp, sizeof(int) * SIZE);
    hipMemcpy(&(d_test->data), &d_tmp, sizeof(int *), hipMemcpyHostToDevice);

    init<<<1, SIZE>>> (d_test);

    hipMemcpy(t->data, d_tmp, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_test);
    hipFree(d_tmp);

    delete t;
}


int main()
{
    for (unsigned i = 0; i < 10; ++i)
        initTest();

    return 0;
}
